#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "common.h"
#include "all_reduce.h"
#include "collectives.h"

#define UNROLL 4

#if NCCL_OP == 0
IMPL_COLL2(ncclAllReduce, sum,  FuncSum,  ncclCollAllReduce, ncclSum);
#elif NCCL_OP == 1
IMPL_COLL2(ncclAllReduce, prod, FuncProd, ncclCollAllReduce, ncclProd);
#elif NCCL_OP == 2
IMPL_COLL2(ncclAllReduce, min,  FuncMin,  ncclCollAllReduce, ncclMin);
#elif NCCL_OP == 3
IMPL_COLL2(ncclAllReduce, max,  FuncMax,  ncclCollAllReduce, ncclMax);
#endif
