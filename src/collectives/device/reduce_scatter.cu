#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 * Modifications Copyright (c) 2019 Advanced Micro Devices, Inc. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "common.h"
#include "reduce_scatter.h"
#include "collectives.h"

#define UNROLL 4

#if NCCL_OP == 0
IMPL_COLL2(ncclReduceScatter, sum,  FuncSum,  ncclCollReduceScatter, ncclSum);
#elif NCCL_OP == 1
IMPL_COLL2(ncclReduceScatter, prod, FuncProd, ncclCollReduceScatter, ncclProd);
#elif NCCL_OP == 2
IMPL_COLL2(ncclReduceScatter, min,  FuncMin,  ncclCollReduceScatter, ncclMin);
#elif NCCL_OP == 3
IMPL_COLL2(ncclReduceScatter, max,  FuncMax,  ncclCollReduceScatter, ncclMax);
#endif
