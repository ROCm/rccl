#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2021, NVIDIA CORPORATION. All rights reserved.
 * Modifications Copyright (c) 2019-2021 Advanced Micro Devices, Inc. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "devcomm.h"
#include "collectives.h"
#include "common.h"

__shared__ ncclShmemData ncclShmem;

#if defined(__HIP_PLATFORM_HCC__) || defined(__HCC__) || defined(__HIPCC__)
#else
#define NCCL_FUNC5(func, algo, devredop, type, nullify) \
  MACRO_IF(nullify, nullptr, NCCL_FUNC_NAME(func, algo, LL,     devredop, type)), \
  MACRO_IF(nullify, nullptr, NCCL_FUNC_NAME(func, algo, LL128,  devredop, type)), \
  MACRO_IF(nullify, nullptr, NCCL_FUNC_NAME(func, algo, SIMPLE, devredop, type))

#define NCCL_FUNC4(func, devredop, type, nullify) \
  NCCL_FUNC5(func, TREE,    devredop, type, nullify), \
  NCCL_FUNC5(func, RING,    devredop, type, nullify), \
  NCCL_FUNC5(func, COLLNET, devredop, type, nullify)

#if defined(__CUDA_BF16_TYPES_EXIST__)
// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(func, devredop, nullForFloat) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, uint8_t, 0), \
  NCCL_FUNC4(func, devredop, int32_t, 0), \
  NCCL_FUNC4(func, devredop, uint32_t, 0), \
  NCCL_FUNC4(func, devredop, int64_t, 0), \
  NCCL_FUNC4(func, devredop, uint64_t, 0), \
  NCCL_FUNC4(func, devredop, half, nullForFloat), \
  NCCL_FUNC4(func, devredop, float, nullForFloat), \
  NCCL_FUNC4(func, devredop, double, nullForFloat), \
  NCCL_FUNC4(func, devredop, __hip_bfloat16, nullForFloat)
#define NCCL_FUNCS3B(func, devredop) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0)
#else
// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(func, devredop, nullForFloat) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, uint8_t, 0), \
  NCCL_FUNC4(func, devredop, int32_t, 0), \
  NCCL_FUNC4(func, devredop, uint32_t, 0), \
  NCCL_FUNC4(func, devredop, int64_t, 0), \
  NCCL_FUNC4(func, devredop, uint64_t, 0), \
  NCCL_FUNC4(func, devredop, half, nullForFloat), \
  NCCL_FUNC4(func, devredop, float, nullForFloat), \
  NCCL_FUNC4(func, devredop, double, nullForFloat)
#define NCCL_FUNCS3B(func, devredop) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0)
#endif

// Must be consistent with ncclRedOp_t
#define NCCL_FUNCS2A(func) \
  NCCL_FUNCS3A(func, Sum,        /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, Prod,       /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, Max,        /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, Min,        /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, PreMulSum,  /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, SumPostDiv, /*nullForFloat=*/1)

#define NCCL_FUNCS2B(func) \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum)

// Must be consistent with the ncclFuncSet enum
__device__ ncclKern_t ncclFuncs[1+ncclNumTypes+NCCL_NUM_FUNCTIONS*ncclNumDevRedOps*ncclNumTypes*NCCL_NUM_ALGORITHMS*NCCL_NUM_PROTOCOLS] = {
// Don't try to initialize the host shadow copy of this device-side global
// variable. There is no host pointer to a device-side function, which
// confuses clang. This will be fixed in the next clang release.
#if __CUDA_ARCH__
  NCCL_FUNC_NAME(SendRecv, RING, SIMPLE, Sum, int8_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, int8_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, uint8_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, int32_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, uint32_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, int64_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, uint64_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, half),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, float),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, double),
  #if defined(__CUDA_BF16_TYPES_EXIST__)
    NCCL_ONERANK_REDUCE_NAME(PreMulSum, __hip_bfloat16),
  #endif
  NCCL_FUNCS2B(Broadcast),
  NCCL_FUNCS2A(Reduce),
  NCCL_FUNCS2B(AllGather),
  NCCL_FUNCS2A(ReduceScatter),
  NCCL_FUNCS2A(AllReduce)
#endif
};
#endif

// Workaround for https://reviews.llvm.org/D55580
__device__ void ncclWorkaroundClangD55580() {}
